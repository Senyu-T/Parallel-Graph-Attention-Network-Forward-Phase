#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <ctype.h>
#include <math.h>
#include "gat_cuda.h"

#define LBLK 32

#define RM(r, c, width) ((r) * (width) + (c))

static inline int updiv(int n, int d) {
    return (n+d-1)/d;
}

__global__ void
cudaBlockKernel(int nnode, int in, int total_out, double *in_features, double *weights, double *out_features){
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    int bi = threadIdx.y;
    int bj = threadIdx.x;

    __shared__ float subA[LBLK * LBLK];
    __shared__ float subB[LBLK * LBLK];
    float sum = 0;

    for (int k = 0; k < in; k += LBLK) {
        subA[RM(bi,bj,LBLK)] = in_features[RM(i,k+bj,in)];
        subB[RM(bi,bj,LBLK)] = weights[RM(k+bi,j,total_out)];

        __syncthreads();

        for (int bk = 0; bk < LBLK; bk++) {
            sum += subA[RM(bi,bk,LBLK)] * subB[RM(bk,bj,LBLK)];
        }

        __syncthreads();
    }
    out_features[RM(i,j,total_out)] = sum;
}

//out_feature is nheads*out
void cudaMultMatrix(int nnode, int in, int total_out, double *in_features,
        double *weights, double *out_features){

    dim3 threadsPerBlock(LBLK, LBLK);
    dim3 blocks(updiv(total_out, LBLK), updiv(nnode, LBLK));
    cudaBlockKernel<<<blocks, threadsPerBlock>>>(nnode, in, total_out, in_features, weights, out_features);
}

__global__ void
cudaLinearlrKernel(int nnode, int nheads, int out, double *linear, double *a, double *linear_lr){
    int i = blockIdx.y * blockDim.y + threadIdx.y; //nid
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    int hid = blockIdx.x;

    double sum = 0;
    for (int k=0; k<out; k++){
        int l_idx = i * nheads * out + hid * out + k;
        int a_idx = hid * out * 2 + k + threadIdx.x * out;
        sum += linear[l_idx] * a[a_idx];
    }
    linear_lr[i * nheads * 2 + j] = sum;

}

void cudaComputeLR(int nnode, int nheads, int out, double *linear, double *a, double *linear_lr){

    dim3 threadsPerBlock(2, LBLK * 2);
    dim3 blocks(nheads, updiv(nnode, LBLK * 2));
    cudaLinearlrKernel<<<blocks, threadsPerBlock>>>(nnode, nheads, out, linear, a, linear_lr);
}


__global__ void
    cudaLReluKernel(int nnode, int nhead, int out, double *linear_lr, int *adj, double *relu_matrix, double *relu_sum){
    int i = blockIdx.y * blockDim.y + threadIdx.y; //nid
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    int hid = blockIdx.x;
    int nei = threadIdx.x;//j in adj matrix

    if (adj[i*nnode + nei]){
        double left = linear_lr[i*2*nhead+hid*2];
        double right = linear_lr[nei*2*nhead+hid*2+1];
        double r = 0;
        if (left+right < 0){
            r = ALPHA * (left+right);
        }else{
            r = left + right;
        }
        relu_matrix[i*nnode*nhead + j] = exp(r);
        relu_sum[i*nnode*nhead + j] = exp(r);
    }

    __syncthreads();

    for (int offset=nnode/2; offset>=1; offset/=2){
        if (nei<offset){
            relu_sum[i*nnode*nhead+j] = relu_sum[i*nnode*nhead+j] + relu_sum[i*nnode*nhead+j+offset];
        }
        __syncthreads();
    }

    relu_matrix[i*nnode*nhead + j] = relu_matrix[i*nnode*nhead + j] / relu_sum[i*nnode*nhead+hid*nnode];
}

void cudaComputeLRelu(int nnode, int nhead, int out, double *linear_lr, int *adj, double *relu_matrix, double *relu_sum){
    //every thread correpsonds to one entry in the adj matrix
    dim3 threadsPerBlock(nnode, 1);
    dim3 blocks(nhead, nnode);
    cudaLReluKernel<<<blocks, threadsPerBlock>>>(nnode, nhead, out, linear_lr, adj, relu_matrix, relu_sum);
}


__global__ void
cudaComputeNewEmbedding(int nnode, int nhead, int out, double *relu_matrix, double *multi_new_embedding, int *neighbor,
                             int *neighbor_start, double *linear){
    int i = blockIdx.y * blockDim.y + threadIdx.y; //nid

    int hid = blockIdx.x;
    int fid = threadIdx.x;

    int nnid_s = neighbor_start[i];
    int nnid_e = neighbor_start[i + 1];

    for (int nnid = nnid_s; nnid < nnid_e; nnid++) {
        int nei = neighbor[nnid];
        multi_new_embedding[i*out*nhead + hid * out + fid] +=
                relu_matrix[i*nnode*nhead+hid*nnode+nei] * linear[nei*nhead*out+hid*out+fid];
    }

}

void cudaNewEmbedding(int nnode, int nhead, int out, double *relu_matrix, double *mult_new_bedding, int *neighbor,
        int *neighbor_start, double *linear){
    //every thread maps to an entry in the new embedding matrix
    dim3 threadsPerBlock(out, 1);

    //every block corresponds to one node in one head
    dim3 blocks(nhead, nnode);
    cudaComputeNewEmbedding<<<blocks, threadsPerBlock>>>(nnode, nhead, out, relu_matrix, mult_new_bedding, neighbor, neighbor_start, linear);


}


// forward for one layer
void forward(layer_t *L, graph_t *g) {
    int nnode = g->nnode;
    int nedge = g->nedge;
    int nhead = L->num_heads;
    int *neighbor = g->neighbor;
    int *neighbor_start = g->neighbor_start;
    int out = L->params[0]->out_feature;
    int in = L->params[0]->in_feature;

//Step 1, compute h*W, all heads are computed together
    double *device_linear;
    double *device_features;
    double *device_weights;

    double *features = g->features;
    double *weights = L->weights;
    hipMalloc((void**)&device_features, nnode * in * sizeof(double));
    hipMalloc((void**)&device_weights, in * out * nhead * sizeof(double));
    hipMalloc((void**)&device_linear, nnode * nhead * out * sizeof(double));

    hipMemcpy(device_features, features, nnode * in * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_weights, weights, in * out * nhead * sizeof(double), hipMemcpyHostToDevice);

    cudaMultMatrix(nnode, in, out*nhead, device_features, device_weights, device_linear);

    hipDeviceSynchronize();

    //computer left and right value for every node, every head
    double *a = L->a;
    double *device_a;
    double *device_linear_lr; //nnode * (2 *nhead)

    hipMalloc((void**)&device_a, 2 * out * nhead * sizeof(double));
    hipMalloc((void**)&device_linear_lr, 2 * nhead * nnode * sizeof(double));
    hipMemcpy(device_a, a, 2 * out * nhead * sizeof(double), hipMemcpyHostToDevice);

    cudaComputeLR(nnode, nhead, out, device_linear, device_a, device_linear_lr);
    hipDeviceSynchronize();

    //Step 2: for every edge apply leakyRelu, then compyre alpha_{ij}
    int *device_adj;
    double *device_relu_matrix;  //nnode * (nnode * nhead)
    double *device_relu_sum; //nnode * (nnode * nhead)
    hipMalloc((void**)&device_adj, nnode * nnode  * sizeof(int));
    hipMemcpy(device_adj, g->adj, nnode * nnode * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&device_relu_matrix, nnode * nnode * nhead * sizeof(double));
    hipMalloc((void**)&device_relu_sum, nnode * nnode * nhead * sizeof(double));

    cudaComputeLRelu(nnode, nhead, out, device_linear_lr, device_adj, device_relu_matrix, device_relu_sum);

    double *relu_matrix = (double *)calloc(sizeof(double), nnode * nnode * nhead);
    hipDeviceSynchronize();

    //Step 3: compute new embedding
    double *device_mult_new_embedding;
    int *device_neighbor;
    int *device_neighbor_start;
    hipMalloc((void**)&device_mult_new_embedding, nnode * nhead * out * sizeof(double));
    hipMalloc((void**)&device_neighbor, (nnode + 2 * nedge) * sizeof(int));
    hipMalloc((void**)&device_neighbor_start, (nnode + 1) * sizeof(int));

    hipMemcpy(device_neighbor, neighbor, (nnode + 2 * nedge) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_neighbor_start, neighbor_start, (nnode + 1) * sizeof(int), hipMemcpyHostToDevice);

    cudaNewEmbedding(nnode, nhead, out, device_relu_matrix, device_mult_new_embedding, device_neighbor,
            device_neighbor_start, device_linear);

    double *multi_new_embedding = (double *)calloc(sizeof(double), nnode * nhead * out);

    hipMemcpy(multi_new_embedding, device_mult_new_embedding, nnode * nhead * out * sizeof(double), hipMemcpyDeviceToHost);

    g->features = multi_new_embedding;
    g->nfeature = out * nhead;
}

/* utility functions */
double lrelu(double x, double alpha) {
    return x < 0 ? alpha * x : x;
}

// concatenation, a, b or of equal size
double *concat_weights(double *a, double *b, int size) {
    double *concat = (double *)calloc(sizeof(double), 2 * size);
    memcpy(concat, a, size * sizeof(double));
    memcpy(concat + size, b, size * sizeof(double));
    return concat;
}



